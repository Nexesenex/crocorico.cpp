#include "hip/hip_runtime.h"
//
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "iqk_mmvq.cuh"

#include "vecdotq.cuh"

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs);

//  Reminder:
//    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
//    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
//    constexpr int vdr = get_vdr_mmvq(type);

namespace {
template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__((ncols_y <= 4 ? 4 : 2)*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__global__ void iqk_mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, const int64_t row_size) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int nwarps              = ncols_y <= 4 ? 4 : 2;
    constexpr int rows_per_cuda_block = ncols_y == 1 ? 1 : 2;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda((const void *)((const char *)vx + (row0 + i)*row_size),
                    &y[j*blocks_per_col_y + kby], kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            dst[j*nrows_dst + row0 + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda>
void iqk_mul_mat_vec_q_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    GGML_ASSERT(ncols_x % ggml_blck_size(type) == 0);
    //GGML_ASSERT(ncols_y <= MMVQ_MAX_BATCH_SIZE);

    int id = ggml_cuda_get_device();

    int64_t nwarps = 1;
    int64_t rows_per_cuda_block = 1;

    if (ggml_cuda_info().devices[id].cc < CC_RDNA2) { // NVIDIA and AMD older than RDNA2
        switch(ncols_y) {
            case 1:
                nwarps = 4;
                rows_per_cuda_block = 1;
                break;
            case 2:
            case 3:
            case 4:
                nwarps = 4;
                rows_per_cuda_block = 2;
                break;
            case 5:
            case 6:
            case 7:
            case 8:
                nwarps = 2;
                rows_per_cuda_block = 2;
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
    }
    const int64_t nblocks = (nrows_x + rows_per_cuda_block - 1) / rows_per_cuda_block;
    const dim3 block_nums(nblocks, 1, 1);
    const dim3 block_dims(WARP_SIZE, nwarps, 1);

    const int64_t row_size = ggml_row_size(type, ncols_x);

    switch (ncols_y) {
        case 1:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 1><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 2:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 2><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 3:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 3><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 4:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 4><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 5:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 5><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 6:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 6><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 7:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 7><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        case 8:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 8><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}



} // namespace

void mul_mat_vec_iq2_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K, VDR_IQ2_K_Q8_1_MMVQ, vec_dot_iq2_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq3_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_K, VDR_IQ3_K_Q8_1_MMVQ, vec_dot_iq3_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq4_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_K, VDR_IQ4_K_Q8_1_MMVQ, vec_dot_iq4_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq4_ks_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KS, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq4_ks_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq4_kss_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KSS, VDR_IQ4_KSS_Q8_1_MMVQ, vec_dot_iq4_kss_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq2_ks_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KS, VDR_IQ2_KS_Q8_1_MMVQ, vec_dot_iq2_ks_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq2_kt_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KT, VDR_IQ2_KS_Q8_1_MMVQ, vec_dot_iq2_kt_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq5_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_K, VDR_IQ5_K_Q8_1_MMVQ, vec_dot_iq5_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq6_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ6_K, VDR_IQ6_K_Q8_1_MMVQ, vec_dot_iq6_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq1_bn_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_BN, 1, vec_dot_iq1_bn_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq2_bn_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_BN, 1, vec_dot_iq2_bn_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}
